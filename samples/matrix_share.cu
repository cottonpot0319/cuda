#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 1600

struct Matrix{
    int width;
    int height;
    int stride;
    double* elements;
};

__device__ 
Matrix GetSubMatrix(Matrix A, int row, int col)
{
    Matrix Asub;
    Asub.width  = BLOCK_SIZE;
    Asub.height = BLOCK_SIZE;
    Asub.stride = A.stride;
    Asub.elements = &A.elements[A.stride*BLOCK_SIZE*row+BLOCK_SIZE*col];

    return Asub;
}

__global__ 
void matrixMulShared(Matrix A, Matrix B, Matrix C)
{
    int row = blockIdx.y*blockDim.y+threadIdx.y;
    int col = blockIdx.x*blockDim.x+threadIdx.x;

    if(row < C.height && col < C.width){
        int brow = blockIdx.y;
        int bcol = blockIdx.x;

        Matrix Csub = GetSubMatrix(C, brow, bcol);

        int trow = threadIdx.y;
        int tcol = threadIdx.x;

        float x = 0.0f;
        for(int l = 0; l < (A.width+BLOCK_SIZE-1)/BLOCK_SIZE; ++l){
            Matrix Asub = GetSubMatrix(A, brow, l);
            Matrix Bsub = GetSubMatrix(B, l, bcol);

            __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
            __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

            // サブ行列の内容をシェアードメモリへ
            As[trow][tcol] = Asub.elements[trow*Asub.stride+tcol];
            Bs[trow][tcol] = Bsub.elements[trow*Bsub.stride+tcol];

            // 他のスレッドがシェアードへの書き込みを終了するのを待つ
            __syncthreads();

            for(int k = 0; k < BLOCK_SIZE; ++k){
                x += As[trow][k]*Bs[k][tcol];
            }

            // 次の反復でサブ行列が書き換えられるため，ここで処理待ち
            __syncthreads();
        }

        Csub.elements[trow*Csub.stride+tcol] = x;
    }
}

int main(int argc, char** argv){
    //結果書き込み用ファイルのオープン
    //FILE *fp=fopen("result.txt","w");

    //ホスト側の行列の定義(サイズはブロックサイズの倍数に設定)
    Matrix hA, hB, hC;
    hA.height = hC.height = 3 * BLOCK_SIZE;
    hA.width  = hB.height = 4 * BLOCK_SIZE;
    hB.width  = hC.width  = 5 * BLOCK_SIZE;
    hA.elements = new double[hA.width * hA.height];
    hB.elements = new double[hB.width * hB.height];
    hC.elements = new double[hC.width * hC.height];
    for(int i = 0; i < hA.height*hA.width; i++) hA.elements[i] = 1.0;
    for(int i = 0; i < hB.height*hB.width; i++) hB.elements[i] = 2.0;

    //デバイス側のメモリ確保とデータ転送
    Matrix dA, dB, dC;
    dA.width = hA.width;    dA.height = hA.height;
    dB.width = hB.width;    dB.height = hB.height;
    dC.width = hC.width;    dC.height = hC.height;
    int size;
    // デバイスメモリの確保とホストからの転送
    size = dA.width*dA.height*sizeof(double);
    hipMalloc((void**)&dA.elements, size);
    hipMemcpy(dA.elements, hA.elements, size, hipMemcpyHostToDevice);
    size = dB.width*dB.height*sizeof(double);
    hipMalloc((void**)&dB.elements, size);
    hipMemcpy(dB.elements, hB.elements, size, hipMemcpyHostToDevice);
    size = dC.width*dC.height*sizeof(double);
    hipMalloc((void**)&dC.elements, size);

    //カーネルの実行
    dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid((dC.width+block.x-1)/block.x, (dC.height+block.y-1)/block.y);
    matrixMulShared<<<grid, block >>>(dA, dB, dC);
    // カーネル実行エラーのチェック
    //cutilCheckMsg("Kernel execution failed");

    // デバイスからホストへ結果を転送
    size = dC.width*dC.height*sizeof(double);
    hipMemcpy(hC.elements, dC.elements, size, hipMemcpyDeviceToHost);

    //タイマーを作成して計測開始
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    //タイマーを停止しかかった時間を表示
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0.0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("time: %8.3f ms\n", milliseconds);

    //計算結果
    //for(int i = 0; i < hC.height*hC.width; i++) std::cout << hC.elements[i] << std::endl; 

    // デバイスメモリ解放
    hipFree(dA.elements);
    hipFree(dB.elements);
    hipFree(dC.elements);
    // ホストメモリ解放
    delete [] hA.elements;
    delete [] hB.elements;
    delete [] hC.elements;
    //fclose(fp);

    //終了処理
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return 0;
}
