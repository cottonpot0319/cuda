#include <stdio.h>
#include <hip/hip_runtime.h>

#include "diff2d_kernel.cu"

//系のサイズ 100 X 100のグリッド上で拡散方程式を解く
const int X=100;
const int Y=100;

int main( int argc, char** argv){
    //デバイスの初期化
    //CUT_DEVICE_INIT(argc, argv);

    //結果書き込み用ファイルのオープン
    FILE *fp=fopen("result.txt","w");

    //タイマーを作成して計測開始
    //unsigned int timer = 0;
    //cutCreateTimer(&timer);
    //cutStartTimer(timer);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    //メインメモリ上にfloat型のデータをX*Y個生成する
    float* h_idata = (float*) malloc(sizeof( float) * X*Y);
    //初期条件をセット
    for( int i = 0; i < X; i++) 
        for( int j = 0; j < X; j++)
            if((i-X/2)*(i-X/2)+(j-Y/2)*(j-Y/2)<10*10)
                h_idata[i*Y+j] = 1;
            else
                h_idata[i*Y+j] = 0;

    //デバイス上（ビデオカードのこと）にも同じくfloat型X*Y個分のメモリを確保する
    float* d_idata;
    hipMalloc((void**) &d_idata, sizeof( float) * X*Y);
    //デバイス上（ビデオカードのこと）にfloat型X*Y個分の作業用メモリを確保する
    float* d_idata2;
    hipMalloc((void**) &d_idata2, sizeof( float) * X*Y);

    //ブロック数を増やして並列度を上げる
    dim3  grid( 16, 1, 1);
    dim3  threads(256, 1, 1);
    
    //メインメモリからデバイスのメモリにデータを転送する
    hipMemcpy( d_idata, h_idata, sizeof( float) * X*Y , hipMemcpyHostToDevice);

    for (int t=0;t<100;t++){
        for (int n=0;n<10;n++){
            //ここでGPUを使った計算が行われる
            diff2dKernel<<< grid, threads>>>( d_idata, d_idata2,X,Y);
            //作業用領域から書き戻す
            hipMemcpy( d_idata, d_idata2, sizeof( float) * X*Y, hipMemcpyDeviceToDevice);
        }
        //デバイスからメインメモリ上に実行結果をコピー
        hipMemcpy( h_idata, d_idata, sizeof( float) * X*Y, hipMemcpyDeviceToHost);
        //実行結果を表示
        for (int i=0;i<X;i+=2){
            for (int j=0;j<Y;j+=2){
                fprintf(fp,"%f\t",h_idata[i*Y+j]);
            }
        }
        fprintf(fp,"\n");
    }
    //タイマーを停止しかかった時間を表示
    //cutStopTimer(timer);
    //printf("Processing time: %f (ms)\n", cutGetTimerValue(timer));
    //cutDeleteTimer(timer);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0.0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("time: %8.2f ms\n", milliseconds);

    //各種メモリを解放
    free(h_idata);
    hipFree(d_idata);
    hipFree(d_idata2);
    fclose(fp);
    //終了処理
    //CUT_EXIT(argc, argv);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return 0;
}
